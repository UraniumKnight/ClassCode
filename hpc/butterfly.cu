#include<stdio.h>
#include <hip/hip_runtime.h>

// Forward Declarations
#define BLOCKSIZE 1024

void printArray(int k);
__global__ void add(int d_a[], int *d_answer);

int* a;
int answer;

 int main(){
   hipError_t err;

   int deviceCount;
   err = hipGetDeviceCount(&deviceCount);
   printf("Device count: %s\n",hipGetErrorString(err));
   printf("There are %d devices\n", deviceCount);

   err = hipSetDevice(0);
   printf("Device selection: %s\n",hipGetErrorString(err));

   int N = 1024;
   a = (int*)malloc(N * sizeof(int));

  // Fill the array
  int i;			/* counter */
  for(i = 0; i < N; i++)
    a[i] = rand() % 23;

  printArray(20);

  // Allocate space on the GPU
  int* d_Array;			/* d_ means "device" */
  int* d_answer;
  err = hipMalloc(&d_Array, N * sizeof(int));
  printf("Malloc device rules: %s\n",hipGetErrorString(err));
  err = hipMalloc(&d_answer, sizeof(long));
  printf("Malloc device rules: %s\n",hipGetErrorString(err));


  // Copy the array to the card
  // destination, then source
  err = hipMemcpy(d_Array, a, N * sizeof(int), hipMemcpyHostToDevice);
  printf("cuda memory error: %s\n",hipGetErrorString(err));
  err = hipMemcpy(d_answer, &answer, sizeof(int), hipMemcpyHostToDevice);
  printf("cuda memory error: %s\n",hipGetErrorString(err));

  // Set up the kernel
  int blockSize = BLOCKSIZE;
  int numBlocks = 1;
  dim3 dimGrid(numBlocks);
  dim3 dimBlock(blockSize);

  // Launch the kernel
  add <<< dimGrid, dimBlock >>> (d_Array, d_answer);

  // Retrieve the results from the card
  err = hipMemcpy(&answer, d_answer, sizeof(int), hipMemcpyDeviceToHost);
  printf("cuda memory error: %s\n",hipGetErrorString(err));
  err = hipMemcpy(a, d_Array, N*sizeof(int), hipMemcpyDeviceToHost);
  printf("cuda memory error: %s\n",hipGetErrorString(err));

  // Inspect the results.
  printf("%i\n", answer);
  printArray(20);
}

void printArray(int k){
  int i;
  for(i = 0; i < k; i++)
    printf("%d ", a[i]);
  printf("\n");
}


__global__ void add(int d_a[], int *d_answer){
  __shared__ int a[BLOCKSIZE];

  a[threadIdx.x] = d_a[threadIdx.x];
  __syncthreads();

  for (int i = 0; i < log2f(blockDim.x); i++){
    int idx = threadIdx.x;
    int neighbor = idx ^ (1<<i);
    int his = a[neighbor];
    int my = a[idx];
    int holder = my + his;
    __syncthreads();
    a[idx] = holder;
    __syncthreads();
  }
  *d_answer = a[threadIdx.x];
}
