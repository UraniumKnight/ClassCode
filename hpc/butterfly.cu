#include "hip/hip_runtime.h"
#include<stdio.h>
#include <hip/hip_runtime.h>

// Forward Declarations
void printArray(int k);
__global__ void add(int d_a[], int d_answer);

int* a;

 int main(){
   hipError_t err;

   int deviceCount;
   err = hipGetDeviceCount(&deviceCount);
   printf("Device count: %s\n",hipGetErrorString(err));
   printf("There are %d devices\n", deviceCount);

   err = hipSetDevice(0);
   printf("Device selection: %s\n",hipGetErrorString(err));

   int N = 1024;
   a = (int*)malloc(N * sizeof(int));

  // Fill the array
  int i;			/* counter */
  for(i = 0; i < N; i++)
    a[i] = rand() % 21;

  printArray(20);

  // Allocate space on the GPU
  int* d_Array;			/* d_ means "device" */
  int d_answer;
  err = hipMalloc(&d_Array, N * sizeof(int));
  printf("Malloc device rules: %s\n",hipGetErrorString(err));
  err = hipMalloc(&d_answer, sizeof(int));
  printf("Malloc device rules: %s\n",hipGetErrorString(err));


  // Copy the aray to the card
  // destination, then source
  hipMemcpy(d_Array, a, N * sizeof(int), hipMemcpyHostToDevice);

  // Set up the kernel
  int blockSize = 1024;
  int numBlocks = 1;
  dim3 dimGrid(numBlocks);
  dim3 dimBlock(blockSize);

  // Launch the kernel
  add <<< dimGrid, dimBlock >>> (d_Array, d_answer);

  // Retrieve the results from the card
  int answer;
  hipMemcpy(&answer, d_answer, sizeof(int), hipMemcpyDeviceToHost);

  // Inspect the results.
  print answer;
}

void printArray(int k){
  int i;
  for(i = 0; i < k; i++)
    printf("%d ", a[i]);
  printf("\n");
}


__global__ void add(int d_a[], int d_answer){
  __shared__ int a[blockDim.x];
  __shared__ int sum;

  a[threadIdx.x] = d_a[threadIdx.x];
  __syncthreads();

  for (int i = 0; i < log(blockDim.x) / log(2); i++){
    int idx = threadIdx.x;
    int neighbor = idx ^ (1<<i);
    int his = a[neighbor];
    int my = a[idx];
    int holder = my + his;
    __syncthreads();
    a[idx] = holder;
    __syncthreads();
  }
  d_answer = a[threadIdx.x];
}


