#include<stdio.h>
#include <time.h> 
#include <hip/hip_runtime.h>

// Forward Declarations
#define BLOCKSIZE 1024
#ifndef Nsize
#define Nsize 1024
#endif

void printArray(int k);
__global__ void add(int d_a[], int *d_answer);

int* a;
int answer;

 int main(){
   hipError_t err;

   int deviceCount;
   err = hipGetDeviceCount(&deviceCount);
   printf("Device count: %s\n",hipGetErrorString(err));
   printf("There are %d devices\n", deviceCount);

   err = hipSetDevice(0);
   printf("Device selection: %s\n",hipGetErrorString(err));

   a = (int*)malloc(Nsize * sizeof(int));

  // Fill the array
  int i;			/* counter */
  time_t t;
  //srand((unsigned) time(&t));
  for(i = 0; i < Nsize; i++)
    a[i] = rand() % 23;

  printArray(Nsize);

  // Allocate space on the GPU
  int* d_Array;			/* d_ means "device" */
  int* d_answer;
  err = hipMalloc(&d_Array, Nsize * sizeof(int));
  printf("Malloc device rules: %s\n",hipGetErrorString(err));
  err = hipMalloc(&d_answer, sizeof(long));
  printf("Malloc device rules: %s\n",hipGetErrorString(err));


  // Copy the array to the card
  // destination, then source
  err = hipMemcpy(d_Array, a, Nsize * sizeof(int), hipMemcpyHostToDevice);
  printf("cuda memory error: %s\n",hipGetErrorString(err));
  err = hipMemcpy(d_answer, &answer, sizeof(int), hipMemcpyHostToDevice);
  printf("cuda memory error: %s\n",hipGetErrorString(err));

  // Set up the kernel
  int blockSize = BLOCKSIZE;
  int numBlocks = 1;
  dim3 dimGrid(numBlocks);
  dim3 dimBlock(blockSize);

  // Launch the kernel
  add <<< dimGrid, dimBlock >>> (d_Array, d_answer);

  // Retrieve the results from the card
  err = hipMemcpy(&answer, d_answer, sizeof(int), hipMemcpyDeviceToHost);
  printf("cuda memory error: %s\n",hipGetErrorString(err));
  err = hipMemcpy(a, d_Array, Nsize*sizeof(int), hipMemcpyDeviceToHost);
  printf("cuda memory error: %s\n",hipGetErrorString(err));

  // Inspect the results.
  printf("%i\n", answer);
  printArray(20);
}

void printArray(int k){
  int i;
  for(i = 0; i < k; i++)
    printf("%d ", a[i]);
  printf("\n");
}


__global__ void add(int d_a[], int *d_answer){
  int idx = threadIdx.x;
  if(idx >= Nsize){
    return;
  }
  __shared__ int a[BLOCKSIZE];

  a[idx] = d_a[idx];
  __syncthreads();

  for (int i = 0; i < (log2f(BLOCKSIZE)); i++){
    int neighbor = idx ^ (1<<i);
    int his = 0;
    if(neighbor >= Nsize){
      his = 0;
    }
    else{
      his = a[neighbor];
    }
    int my = a[idx];
    int holder = my + his;
    __syncthreads();
    a[idx] = holder;
    __syncthreads();
  }
  *d_answer = a[idx];
}
