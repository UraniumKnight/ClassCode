#include "hip/hip_runtime.h"
#include<stdio.h>
#include <hip/hip_runtime.h>

// Forward Declarations
void printArray(int k);
__global__ void dotprod(double d_a[], double d_b[]);

double *a, *b;
double *result;

int main(){
  hipError_t err;

  int deviceCount;
  err = hipGetDeviceCount(&deviceCount);
  printf("Device count: %s\n",hipGetErrorString(err));
  printf("There are %d devices\n", deviceCount);

  err = hipSetDevice(0);
  printf("Device selection: %s\n",hipGetErrorString(err));

  int N = 100000000;
  // Space on the host for two double arrays
  a = (double*)malloc(N * sizeof(double));
  b = (double*)malloc(N * sizeof(double));
  

  // Fill the arrays
  int i;            /* counter */
  for(i = 0; i < N; i++){
    a[i] = ((double)rand())/rand();
    b[i] = ((double)rand())/rand();
  }
  printArray(20);

  // Allocate space on the GPU
  double* d_Array_a;
  double* d_Array_b;
  double* d_result; 
  err = hipMalloc(&d_Array_a, N * sizeof(double));
  printf("Malloc device rules: %s\n",hipGetErrorString(err));
  err = hipMalloc(&d_Array_b, N * sizeof(double));
  printf("Malloc device rules: %s\n",hipGetErrorString(err));
  err = hipMalloc(&d_result, sizeof(double));
  printf("Malloc device rules: %s\n",hipGetErrorString(err));

  // Copy the aray to the card
  // destination, then source
  err = hipMemcpy(d_Array_a, a, N*sizeof(double), hipMemcpyHostToDevice);
  printf("Memory copy error: %s\n", hipGetErrorString(err));
  err = hipMemcpy(d_Array_b, b, N*sizeof(double), hipMemcpyHostToDevice);
  printf("Memory copy error: %s\n", hipGetErrorString(err));
  err = hipMemcpy(d_result, result, sizeof(double), hipMemcpyHostToDevice);
  printf("Memory copy error: %s\n", hipGetErrorString(err));

  // Set up the kernel
  int blockSize = 512;
  int numBlocks = (N + blockSize - 1)/blockSize;
  dim3 dimGrid(numBlocks);
  dim3 dimBlock(blockSize);

  // Launch the kernel
  dotprod <<< dimGrid, dimBlock >>> (d_Array_a, d_Array_b, result);

  // Retrieve the results from the card
  err = hipMemcpy(result, d_result, N*sizeof(double), hipMemcpyDeviceToHost);
  printf("Memory copy error: %s\n", hipGetErrorString(err));

  // Inspect the results.
  printf("dot product result: %s\n", result);

}

void printArray(int k){
  int i;
  for(i = 0; i < k; i++)
    printf("%d ", a[i]);
  printf("\n");
}

__global__ void dotprod(double d_a[], double d_b[], double r[]){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  double val = d_a[idx] * d_b[idx];
  atomicAdd(&(r[0]), val);
}
